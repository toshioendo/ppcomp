
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define NX 20000
#define NY 20000

float data[2][NY][NX];

/* in microseconds (us) */
double get_elapsed_time(struct timeval *begin, struct timeval *end)
{
    return (end->tv_sec - begin->tv_sec) * 1000000
        + (end->tv_usec - begin->tv_usec);
}

void init()
{
    int x, y;
    int cx = NX/2, cy = 0; /* center of ink */
    int rad = (NX+NY)/8; /* radius of ink */
    
    for(y = 0; y < NY; y++) {
        for(x = 0; x < NX; x++) {
            float v = 0.0;
            if (((x-cx)*(x-cx)+(y-cy)*(y-cy)) < rad*rad) {
                v = 1.0;
            }
            data[0][y][x] = v;
            data[1][y][x] = v;
        }
    }
    return;
}

/* Calculate for one time step */
/* Input: data[t%2], Output: data[(t+1)%2] */
void calc(int nt)
{
    int t, x, y;

    for (t = 0; t < nt; t++) {
        int from = t%2;
        int to = (t+1)%2;

#if 1
        printf("step %d\n", t);
        //fflush(0);
#endif
    
        for (y = 1; y < NY-1; y++) {
            for (x = 1; x < NX-1; x++) {
                data[to][y][x] = 0.2 * (data[from][y][x]
                                        + data[from][y][x-1]
                                        + data[from][y][x+1]
                                        + data[from][y-1][x]
                                        + data[from][y+1][x]);
            }
        }
    }

    return;
}

int  main(int argc, char *argv[])
{
    struct timeval t1, t2;
    int nt = 20; /* number of time steps */
  
    if (argc >= 2) { /* if an argument is specified */
        nt = atoi(argv[1]);
    }

    init();

    gettimeofday(&t1, NULL);

    calc(nt);

    gettimeofday(&t2, NULL);

    {
        double us;
        double gflops;
        int op_per_point = 5; // 4 add & 1 multiply per point

        us = get_elapsed_time(&t1, &t2);
        printf("Elapsed time: %.3lf sec\n", us/1000000.0);
        gflops = ((double)NX*NY*nt*op_per_point)/us/1000.0;
        printf("Speed: %.3lf GFlops\n", gflops);
    }

    return 0;
}
