
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (100)
#define BS (5)

__global__ void add(int *DA, int *DB)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    /* printf("Hello GPU world I'm %d\n", id); */
    DA[i] += DB[i];
    return;
}

int main(int argc, char *argv[])
{
    int i;
    int A[N], B[N];
    int *DA, *DB;

    // Initialize arrays
    for (i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i*2;
    }

    hipMalloc((void **)&DA, sizeof(int)*N);
    hipMalloc((void **)&DB, sizeof(int)*N);

    hipMemcpy(DA, A, sizeof(int)*N, hipMemcpyDefault);
    hipMemcpy(DB, B, sizeof(int)*N, hipMemcpyDefault);

    /* call GPU kernel function with N threads */
    add<<<N/BS, BS>>>(DA, DB);

    hipMemcpy(A, DA, sizeof(int)*N, hipMemcpyDefault);

    hipFree(DB);
    hipFree(DA);

    printf("A[%d]=%d\n", 0, A[0]);
    printf("A[%d]=%d\n", N-1, A[N-1]);

    return 0;
}

